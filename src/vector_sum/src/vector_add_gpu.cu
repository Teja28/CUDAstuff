#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/vector_add_gpu.h"
using namespace std;


void checkCUDAError(hipError_t err) {
	if(err != hipSuccess) {
		cout << "The error is " << hipGetErrorString(hipGetLastError()) << ".";
		cout << endl;
	}
}

__global__ void vectorAddGPU(float *a, float *b, float *c, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < N) {
		c[i] = a[i] + b[i];
	}
}

void vectorAddGPU_wrapper(float *h_a, float *h_b, float h_c[], int N) {
	float *dev_a;
	float *dev_b;
	float *dev_c;
	size_t size = N * sizeof(float);

	float ms0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Allocate GPU memory
	checkCUDAError(hipMalloc(&dev_a, size));
	checkCUDAError(hipMalloc(&dev_b, size));
	checkCUDAError(hipMalloc(&dev_c, size));

	// Copy host data to device
	checkCUDAError(hipMemcpy(dev_a, h_a, size, hipMemcpyHostToDevice));
	checkCUDAError(hipMemcpy(dev_b, h_b, size, hipMemcpyHostToDevice));
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms0, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	cout << "GPU Memory Allocation/Copy to Device: " << ms0 << " ms" << endl;
	float ms = ms0;

	int blockSize;
	int gridSize;
	if(N < 1024) {
		blockSize = N;
		gridSize = 1;
	}
	else {
		blockSize = 1024;
		gridSize = (int)ceil((float)N / blockSize); 
	}

	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventRecord(start1, 0);
	vectorAddGPU<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, N);
	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	float ms1;
	hipEventElapsedTime(&ms1, start1, stop1);
	hipEventDestroy(start1);
	hipEventDestroy(stop1);
	cout << "Computation Time: " << ms1 << " ms" << endl;
	ms += ms1;

	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2, 0);
	checkCUDAError(hipMemcpy(h_c, dev_c, size, hipMemcpyDeviceToHost));
	checkCUDAError(hipFree(dev_a));
	checkCUDAError(hipFree(dev_b));
	checkCUDAError(hipFree(dev_c));
	hipEventRecord(stop2, 0);
	hipEventSynchronize(stop2);
	float ms2;
	hipEventElapsedTime(&ms2, start, stop);
	hipEventDestroy(start2);
	hipEventDestroy(stop2);
	cout << "GPU Memory Deallocation/Copy to Host: " << ms2 << " ms" << endl;
	cout << "Total GPU Time: " << ms << " ms" << endl;
}

// int main() {
// 	cout << "Enter desired vector length: ";
// 	int N;
//     cin >> N;

//     float *a;
//     float *b;
//     float *c;

//     allocMemory(&a, N);
//     allocMemory(&b, N);
//     allocMemory(&c, N);

//     srand(time(NULL));
//     fillVector(a, N);
//     fillVector(b, N);

//     vectorAddGPU_wrapper(a, b, c, N);
	
// 	printVector(c, N);
// }