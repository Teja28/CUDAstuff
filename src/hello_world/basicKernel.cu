
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}


int main( void ) {
    int c;
    int *dev_c;
    
    //Device Memory allocations
    hipError_t err = hipMalloc((void**)&dev_c, sizeof(&dev_c));
    if(err != hipSuccess) {
	   printf("The error is %s\n", hipGetErrorString(err));
    }

    add<<<1,1>>>(2, 7, dev_c);
    
    if(hipPeekAtLastError() != hipSuccess) {
	   printf("The error is %s\n", hipGetErrorString(hipGetLastError()));
    }


    hipError_t err2 = hipMemcpy( &c, dev_c, sizeof(c), hipMemcpyDeviceToHost);
    if(err2 != hipSuccess) {
	   printf("The error is %s\n", hipGetErrorString(err2));
    }


    printf("2 + 7 = %d\n", c);
    hipFree(dev_c);

    return 0;
}
